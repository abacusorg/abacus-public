#include "hip/hip_runtime.h"
/** \file The top-level CUDA code to control the GPUs, particularly in 
the non-blocking mode.

In particular, the preferred non-blocking mode operates several 
independent threads, typically 3 per CUDA Device.  A task operates
on one thread end-to-end, including communication to & from the GPU
and the computation itself.  Each thread controls one CUDA stream.
Because there are multiple streams on each GPU, the GPU can be 
busy computing on one task while communication happens on others.

For the global near-field forces, the work is organized into Sink
Pencils and Source Pencils.  A given cell is part of multiple sink
pencils and hence generates multiple partial accelerations that must
be coadded later.  A task would involve a portion of the sink pencils
as well as the associated source pencils, subdivided so that the 
memory on the GPU is not exhausted. 

For the microstepping, the task involves executing multiple kick-drift-kick
tree-based steps on the GPU, returning the positions and velocities.
The task includes many groups.

Each task therefore has to be prepared on the CPU, assembling a large 
amount of data and the inter-relations needed for the computation.
The GPU task then copies this data to a pinned memory buffer on the
host side and then onto the GPU.  Then the computation kernel is called.
When that is done, the result is copied back to the pinned memory
and then to the destination.

This file contains the functions to set up the GPU threads, 
allocate the pinned memory, and then to run the GPU thread by
receiving tasks and dispensing them to the correct function.

The data model and specifics of the two task use-cases are in 
other files.

Pinned memory is slow to allocate and so we set up a large buffer
for each thread at the beginning of the time slice.  Each task
then partitions up its thread's buffer to match its needs.

This file is compiled separately (hence the headers) and then linked
to the rest of the code, because it needs CUDA.

*/

#include "config.h"
#include "header.cpp"

//#include <x86intrin.h>
#include <cstring>
#include <cstdio>
#include <cassert>
#include <pthread.h>
#include "omp.h"
#include <numaif.h>
#include <atomic>

#include "CudaErrors.cuh"

#ifndef NFRADIUS
#define NFRADIUS 2
#endif

#define WIDTH (2*NFRADIUS +1)

#ifdef DOUBLEPRECISION
    #define FLOAT double
    #define FLOAT4 double4
    #define FLOAT3 double3
    #define RSQRT rsqrt
    #define MIN fmin
    #define ItemSize 2
#else
    #define FLOAT float
    #define FLOAT4 float4
    #define FLOAT3 float3
    #define RSQRT rsqrtf
    #define MIN fminf
    #define ItemSize 1
#endif

#define posstruct FLOAT3
#ifdef COMPUTE_FOF_DENSITY
    #define accstruct FLOAT4
#else
    #define accstruct FLOAT3
#endif

#include "STimer.h"
#include "DeviceFunctions.h"
#include "IncludeGPUKernels.cuh"

void stdlog_hook(int verbosity, const char* str);
#define STDLOG stdlog_hook
#define assertf(_mytest,...) do { \
    if (!(_mytest)) { \
        char logstr[1024];\
        sprintf(logstr, "Failed Assertion: %s\n", #_mytest);\
        STDLOG(0,logstr);\
        fprintf(stderr,"%s",logstr); \
	sprintf(logstr,__VA_ARGS__); \
        STDLOG(0,logstr);\
        assert(0==99); \
    }} while(0)


// Forward declarations that will get linked later
int set_core_affinity(int core_id);

#include "SetInteractionCollection.hh"

int NGPU = 0;		// Number of actual CUDA devices (not threads)

// ===================  The Queue of Tasks =====================

/// We communicate to the queues with a simple FIFO queue. 
/// Tasks go in, and are read out by the QueueWatchers.
/// This class is the item that goes into that queue for
/// the inter-thread communication.
struct GPUQueueTask {
    void *task; ///< The pointer to the task data
    int type;   ///< ==0 signals to kill the Watcher
    	///< ==1 is for SetInteractionCollections
};

#include "tbb/concurrent_queue.h"
tbb::concurrent_bounded_queue<GPUQueueTask> work_queues[MAX_GPUS];

// =============== Code to invoke execution on a SIC ============

// int CurrentGPU = 0;

void GPUPencilTask(void *, int);

/// This submits a single SetInteractionCollection to the GPU queue.
void SetInteractionCollection::GPUExecute(int blocking){
    // This method is invoked on a SIC in order to schedule it for execution.
    // At this point, we are directing work to individual GPUs.

    // Push to the NUMA-appropriate queue
    //int g = CurrentGPU;
    int g = ((j_low + j_high) / 2. / cpd) * NGPU;
    if(j_low == j_high && j_high == cpd)
        g--;
    assert(g < NGPU);

    AssignedDevice = g;  // no meaning for work_queue
    Blocking = blocking;
    
    if(blocking)
        GPUPencilTask(this, AssignedDevice);
    else {
	GPUQueueTask t;
	t.type = 1;
	t.task = (void *)this;
        work_queues[g].push(t);
    }
    //CurrentGPU = (CurrentGPU + 1) %(NGPU * BPD);
}



// =============== GPU Buffers =============================

/// The memory in each GPU is divided into a handful of equal sized
/// buffers, with matching pinned space on the host.  
/// Each buffer is paired with one thread.  Tasks will occupy one
/// buffer.  The memory allocations persist until the thread ends.

struct GPUBuffer {
    uint64 size;	///< The size in bytes
    uint64 sizeWC;	///< The size in bytes of the WC part
    uint64 sizeDef;	///< The size in bytes of the non-WC part
    char * device;	///< The device-side memory
    char * host;	///< The host-side memory allocated as default
    char * hostWC;	///< The host-side memory allocated as write combined
};

hipStream_t * DeviceStreams;	///< The actual CUDA streams, one per thread
GPUBuffer * Buffers; 		///< The pointers to the allocated memory
pthread_t *DeviceThread;	///< The threads!



// ====================== GPU configuration =====================

/// Function to get the number of GPUs
extern "C"
// Use ./configure --with-max-gpus=1 to force 1 GPU
int GetNGPU(){
    int ngpu;
    checkCudaErrors(hipGetDeviceCount(&ngpu));
    if(ngpu > MAX_GPUS)
        ngpu = MAX_GPUS;
    return ngpu;
}

/// Function to get the amount of memory on the GPUs
extern "C" double GetDeviceMemory(){
    int ngpu;
    checkCudaErrors(hipGetDeviceCount(&ngpu));
    double mem = 1e99;
    for(int g = 0; g < ngpu; g++){
        hipDeviceProp_t p;
        checkCudaErrors(hipGetDeviceProperties(&p,g));
        double m = p.totalGlobalMem/1e9;
        mem = min(.8*m,mem);
    }
    return mem;
}


// =============== Setting up the GPUs =========================

// These are the sizes we allocate on the GPU/pinned memory
int MaxSinkBlocks, MaxSourceBlocks;
int MaxNSink, MaxNSource;
size_t MaxSinkSize, MaxSourceSize;

int init = 0;
int BPD; 		// Buffers per device

static std::atomic<uint64> host_alloc_bytes;
static std::atomic<bool> thread_setup_done;

void *QueueWatcher(void *);

// Here is the routine that is called to configure the GPU threads
// and then actually initiate the threads.

// We given the number of GPUs, buffers per GPU, and bytes per buffer.
// Also CPD and the number of particles.
// We use this to compute and return maximum task sizes.

/*
For the Pencil task, the average memory usage:

Memory per Block:
Sink pos and accel:  
	FLOAT3*NFBlockSize bytes of WC memory
	+ accstruct*NFBlockSize bytes of Default memory
Source Pos: FLOAT3*NFBLockSize, but we expect 5x more SourceBlocks
SinkBlockParentPencil: An extra 4 bytes per block.

Sink Pencils: 4+(4+FLOAT)*WIDTH
Source Pencils: 4+4
    This is very small, typically <1%
    We can be conservative and assume this to be per block rather than per cell.
    This slightly neglects the fact that the sources need 4 boundary
    rows, but we just built in a little extra.

In summary:
Per SourceBlock: FLOAT3*NFBlockSize+8/WIDTH of WC memory
Per SinkBlock:   FLOAT3*NFBlockSize+4+FLOAT+4/WIDTH of WC memory
                 accstruct*NFBlockSize of default memory

Assuming that the number of SourceBlocks is WIDTHx SinkBlocks,
this sets the ratio of WC to default memory.  And we can then
return the maximum number of Source and Sink Blocks, so that
the upstream routine can size the SIC to match.

We ignore the fact that the Sink/Source Pencil count may be 
limited by the fraction of CPD**2

*/



/// This launches all of the GPU threads and associated control.
///
/// We will be running multiple GPU threads, usually a few per device.
/// Each thread can only do one task (e.g., a SIC) at a time, and 
/// it controls a single CUDA stream that guarantees sequential 
/// execution of the load, compute, and store cycle.
/// We have multiple threads per GPU so that load/store can be
/// overlapped with compute.
/// 
/// This routine is supplied with the size (in bytes) of each
/// GPU buffer, and it returns estimates on that basis of how
/// big the tasks can be.  This is used in the planning of tasks.

extern "C" void GPUSetup(int cpd, uint64 MaxBufferSize, 
    int numberGPUs, int bufferperdevice, 
    int *ThreadCoreStart, int NThreadCores,
    int *maxsinkblocks, int *maxsourceblocks) {

    if (init != 0) return;
    BPD = bufferperdevice;
    NGPU = numberGPUs;
    int NBuf = BPD*NGPU;
    char logstr[1024];

    // The following estimates are documented above, and assume sizeof(int)=4
    float BytesPerSourceBlockWC = sizeof(FLOAT)*3*NFBlockSize+8.0;
    float BytesPerSinkBlockWC = sizeof(FLOAT)*3*NFBlockSize
    		+(4.0+sizeof(FLOAT))*WIDTH+4.0;
    float BytesPerSinkBlockDef = sizeof(accstruct)*NFBlockSize;

    // Set the levels assuming WIDTH Sources per Sink
    float TotalBytesPerSinkBlock = BytesPerSinkBlockDef+
    	BytesPerSinkBlockWC+WIDTH*BytesPerSourceBlockWC;
    sprintf(logstr, "Bytes per Block = %5.1f+%5.1f+%d*%5.1f = %5.1f\n",
    	BytesPerSinkBlockDef, BytesPerSinkBlockWC, WIDTH, BytesPerSourceBlockWC,
	TotalBytesPerSinkBlock);
    STDLOG(1,logstr);

    // We're splitting the Buffer into WC and Def memory
    float RatioDeftoAll = BytesPerSinkBlockDef/TotalBytesPerSinkBlock;
    assert(RatioDeftoAll<=1.0);  // Guard against screwup

    // This is how many blocks we'll allocate
    MaxSinkBlocks = (MaxBufferSize-1e5)/TotalBytesPerSinkBlock;
    	// Remove 100KB for alignment factors
    MaxSourceBlocks = WIDTH*MaxSinkBlocks;
    *maxsinkblocks = MaxSinkBlocks;
    *maxsourceblocks = MaxSourceBlocks;
    // The number of particles corresponding to these
    MaxSinkSize     = NFBlockSize * MaxSinkBlocks;
    MaxSourceSize   = NFBlockSize * MaxSourceBlocks;

    sprintf(logstr, "Planning for %d sink and %d source blocks, each %d particles\n",
    	MaxSinkBlocks, MaxSourceBlocks, NFBlockSize);
    STDLOG(1,logstr);

    // And then we have storage for the Pencils.
    // Here we pessimistically assume one Pencil per SinkBlock.
    MaxNSink = MaxSinkBlocks;
    MaxNSource = MaxNSink;
    assertf(MaxNSink>2*cpd, "MaxNSink = %d is too small\n", MaxNSink);
    assertf(MaxNSource>(2+WIDTH)*cpd, "MaxNSource = %d is too small\n", MaxNSource);
    // Formally, this one could be slightly larger because of the boundary
    // rows.  However, we neglected this in the Memory estimate, so we'll
    // overflow our buffer if we put it in here.  Fortunately, 1 Pencil
    // per SinkBlock assumption is really very conservative.  Only likely
    // counter-case is very low PPC sims, where one would typically have
    // a small problem that fits very easily on the GPU (i.e., MaxBufferSize
    // will have led to a MaxNSink that is > CPD**2.


    DeviceStreams = new hipStream_t[NBuf];
    Buffers = new GPUBuffer[NBuf];
    DeviceThread = new pthread_t[NBuf];
    // TODO: These appear to never be freed
    
    // Start one thread per buffer
    
    // Assign threads to cores
    // TODO: make this automatic using libnuma
    //int n_socket = numa_available() == -1 ? 1 : (numa_max_node() + 1);
    //sprintf(logstr, "Detected %d sockets/NUMA nodes\n", n_socket);
    //STDLOG(1,logstr);
    //assertf(n_socket > 0, "n_socket %d less than 1\n", n_socket);

    int use_pinned = MaxSinkBlocks >= 10000;  // Pinning is slow, so for very small problems it's faster to use unpinned memory

    if(use_pinned)
        sprintf(logstr, "Allocating pinned memory\n");
    else
        sprintf(logstr, "Allocating host-side memory, but not pinning because this is a small problem\n");
    STDLOG(1,logstr);
    
    for(int g = 0; g < NBuf; g++){
    	Buffers[g].size = MaxBufferSize;
    	Buffers[g].sizeWC = Buffers[g].size*(1.0-RatioDeftoAll);
    	Buffers[g].sizeDef = Buffers[g].size*RatioDeftoAll;

        int buffer_and_core[3];
        int core_start = ThreadCoreStart[g % NGPU];
        int core = -1;
        // If either the core start or the core count are invalid, do not bind this thread to a core
        if(core_start >= 0 && NThreadCores > 0)
            // cycle through cores, but keep each GPU within P.NGPUThreadCores of its starting core
            core = core_start + ((g/NGPU) % NThreadCores);
        
        buffer_and_core[0] = g;
        buffer_and_core[1] = core;
        buffer_and_core[2] = use_pinned;
        if(core >= 0)
            sprintf(logstr, "GPU buffer thread %d (GPU %d) assigned to core %d, use_pinned %d\n", g, g % NGPU, core, use_pinned);
        else
            sprintf(logstr, "GPU buffer thread %d (GPU %d) not bound to a core, use_pinned %d\n", g, g % NGPU, use_pinned);
        STDLOG(0, logstr);
        
        // Start one thread per buffer
        thread_setup_done = 0;
        int p_retval = pthread_create(&(DeviceThread[g]), NULL, QueueWatcher, buffer_and_core);
        assertf(p_retval == 0, "pthread_create failed with value %d\n", p_retval);
        while(!thread_setup_done){}  // only let one thread init at a time
    }
    sprintf(logstr, "Allocated %f MB host-side memory\n", host_alloc_bytes/1024./1024);
    STDLOG(1,logstr);

    init = 1;
}


/// This is the routine to turn off the GPU threads,
/// reset the device, and delete the host-side memory.

void GPUReset(){
    GPUQueueTask t;
    t.type = 0;
    t.task = NULL;
    for(int g = 0; g < NGPU; g++)
    	for (int b = 0; b < BPD; b++) 
        	work_queues[g].push(t);
    for(int g = 0; g < BPD*NGPU; g++)
        assert(pthread_join(DeviceThread[g], NULL) == 0);

    hipDeviceReset();

    delete[] Buffers;
    delete[] DeviceStreams;
    delete[] DeviceThread;
}


// =============== Main code for the GPU thread =================

#define CudaAllocate(ptr,size) checkCudaErrors(hipMalloc((void **)&(ptr), size))

#define PinnedAllocate(ptr,size) if(use_pinned) checkCudaErrors(hipHostAlloc((void **)&(ptr), size, hipHostMallocDefault)); \
                                    else assert(posix_memalign((void **)&(ptr), 64, size) == 0); host_alloc_bytes += size

#define WCAllocate(ptr,size) if(use_pinned) checkCudaErrors(hipHostAlloc((void **)&(ptr), size, hipHostMallocWriteCombined)); \
                                    else assert(posix_memalign((void **)&(ptr), 64, size) == 0); host_alloc_bytes += size

#include "hip/hip_runtime_api.h"

/// Here is the top-level function for each GPU thread.
/// It allocates the space, then monitors a queue to get its work tasks
/// and then invoke the Task function.  When it receives the queue
/// signal to quit, it deletes its buffer memory.
///
/// A wrapper to GPUPencilTask that executes work units from the queue
/// Intended to be executed by several threads/buffers in parallel
/// Our original intention was to let the CUDA stream be our "queue"
/// but the copyback operation is blocking unless we pin memory
/// So this serves as a non-blocking queue.
///
/// By handling buffer memory here, we get a favorable NUMA situation.
/// However, it is also useful to have the thread running on the 
/// correct socket for its PCI slot, so that is passed in as well
/// to run set_core_affinity().

void *QueueWatcher(void *_arg){
	// Parse the device and core args
	// Entry 0 is the assigned buffer number
	// Entry 1 is the assigned core
	// Entry 2 instructs to use pinned memory on the host side
    char logstr[1024]; sprintf(logstr," ");
    int* arg = (int *) _arg;
    int assigned_device = ((int*) arg)[0];
    int n = assigned_device; 		// The buffer number
    int gpu = assigned_device % NGPU;	// The GPU device number
    int assigned_core = ((int*) arg)[1];
    if (assigned_core >= 0)
        set_core_affinity(assigned_core);
    int use_pinned = ((int*) arg)[2];
    sprintf(logstr,"Running GPU thread %d, core %d\n", n, assigned_core); STDLOG(1,logstr);

    checkCudaErrors(hipSetDevice(gpu));
    if (assigned_device < NGPU) {
        // Only run these the first time a GPU is seen
        checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
        checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
    }
    // Initiate the stream
    checkCudaErrors(hipStreamCreateWithFlags(&DeviceStreams[n], hipStreamNonBlocking));
    sprintf(logstr,"GPU stream %d initiated\n", n); STDLOG(1,logstr);

    // Allocate CUDA memory
    CudaAllocate(Buffers[n].device,     Buffers[n].size);
    WCAllocate(Buffers[n].hostWC,       Buffers[n].sizeWC);
    PinnedAllocate(Buffers[n].host,     Buffers[n].sizeDef);
    sprintf(logstr,"GPU thread %d memory allocated\n", n); STDLOG(1,logstr);
    // We make 2/3 of the host pinned memory WriteCombined, which is
    // good for sending data to the GPU.  The other 1/3 is normal, 
    // better for returning the data from the GPU.

    // Attempt some NUMA specifics
    
#ifdef HAVE_LIBNUMA
    // Query the current NUMA node of the allocated buffers
    // We are using the move_pages function purely to query NUMA state, not move anything
    int page = -1;
    move_pages(0, 1, (void **) &(Buffers[n].host), NULL, &page, 0);
    sprintf(logstr, "Host buffer for GPU %d allocated on NUMA node %d on core %d\n", gpu, page, assigned_core);
    move_pages(0, 1, (void **) &(Buffers[n].hostWC), NULL, &page, 0);
    sprintf(logstr, "Host write-combined buffer for GPU %d allocated on NUMA node %d on core %d\n", gpu, page, assigned_core);
    STDLOG(1, logstr);
#endif

    thread_setup_done = 1;  // signal that the next thread can start its setup

    // Main work loop: watch the queue
    while(true){
        GPUQueueTask item;
        work_queues[gpu].pop(item);
        if (item.type == 0)
            break;
        // Each thread (not work unit) is bound to a device

	if (item.type == 1)
	    GPUPencilTask(item.task, assigned_device);
    }
    
    // All done; make sure profiling info is sync'd
    checkCudaErrors(hipStreamSynchronize(DeviceStreams[assigned_device]));
    checkCudaErrors(hipProfilerStop());

    // Free our memory
    checkCudaErrors(hipFree(Buffers[n].device));
    if (use_pinned) {
	checkCudaErrors(hipHostFree(Buffers[n].hostWC));
	checkCudaErrors(hipHostFree(Buffers[n].host));
    } else {
	free(Buffers[n].hostWC);
	free(Buffers[n].host);
    }
    
    return NULL;
}

#undef CudaAllocate
#undef PinnedAllocate
#undef WCAllocate


// ================= Timing the GPUs ==================

// TODO: This may need some adjustment when there's more than one task type.
// Or maybe not?  


/// Timing the GPU is tricky because there are multiple threads 
/// issuing work to it.  So we keep a global counter of all of 
/// active threads, and run a timer when that count is non-zero.
/// That gives a global throughput number.

void CUDART_CB StartThroughputTimer(hipStream_t stream, hipError_t status, void *data){
    assert(pthread_mutex_lock(&SetInteractionCollection::GPUTimerMutex) == 0);
    SetInteractionCollection::ActiveThreads++;
    if (SetInteractionCollection::ActiveThreads == 1)
        SetInteractionCollection::GPUThroughputTimer.Start();
    assert(pthread_mutex_unlock(&SetInteractionCollection::GPUTimerMutex) == 0);
}

void CUDART_CB MarkCompleted( hipStream_t stream, hipError_t status, void *data){
#ifdef CUDADIRECT
    
    assert(pthread_mutex_lock(&SetInteractionCollection::GPUTimerMutex) == 0);
    SetInteractionCollection::ActiveThreads--;
    if (SetInteractionCollection::ActiveThreads == 0)
        SetInteractionCollection::GPUThroughputTimer.Stop();
    assert(pthread_mutex_unlock(&SetInteractionCollection::GPUTimerMutex) == 0);
#endif
}


// ==================  The specific tasks =======================

/// Counting the number of direct interactions performed
__device__ unsigned long long int DI;

__global__ void getDI(unsigned long long int * h_di){
    *h_di = DI;
}

// Here is the code that knows how to execute a single Pencil task.
#include "PencilTask.cu"
