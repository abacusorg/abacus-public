#include "hip/hip_runtime.h"
/** \file The top-level CUDA code to control the GPUs, particularly in 
the non-blocking mode.

In particular, the preferred non-blocking mode operates several 
independent threads, typically 3 per CUDA Device.  A task operates
on one thread end-to-end, including communication to & from the GPU
and the computation itself.  Each thread controls one CUDA stream.
Because there are multiple streams on each GPU, the GPU can be 
busy computing on one task while communication happens on others.

For the global near-field forces, the work is organized into Sink
Pencils and Source Pencils.  A given cell is part of multiple sink
pencils and hence generates multiple partial accelerations that must
be coadded later.  A task would involve a portion of the sink pencils
as well as the associated source pencils, subdivided so that the 
memory on the GPU is not exhausted. 

For the microstepping, the task involves executing multiple kick-drift-kick
tree-based steps on the GPU, returning the positions and velocities.
The task includes many groups.

Each task therefore has to be prepared on the CPU, assembling a large 
amount of data and the inter-relations needed for the computation.
The GPU task then copies this data to a pinned memory buffer on the
host side and then onto the GPU.  Then the computation kernel is called.
When that is done, the result is copied back to the pinned memory
and then to the destination.

This file contains the functions to set up the GPU threads, 
allocate the pinned memory, and then to run the GPU thread by
receiving tasks and dispensing them to the correct function.

The data model and specifics of the two task use-cases are in 
other files.

Pinned memory is slow to allocate and so we set up a large buffer
for each thread at the beginning of the time slice.  Each task
then partitions up its thread's buffer to match its needs.

This file is compiled separately (hence the headers) and then linked
to the rest of the code, because it needs CUDA.

*/

#include "config.h"
#include "header.cpp"


#ifdef __INTEL_COMPILER
#include <x86intrin.h>  // needed for _rdtsc(). only for nvcc using icc with certain gcc versions?
#endif

#include <cstring>
#include <cstdio>
#include <cassert>
#include <pthread.h>
#include "omp.h"

#ifdef HAVE_LIBNUMA
#include <numaif.h>
#endif

#include "CudaErrors.cuh"

#ifndef NFRADIUS
#define NFRADIUS 2
#endif

#define WIDTH (2*NFRADIUS +1)

// TODO: these are NOT using the Abacus-defined float3/double3
// CUDA unfortunately defines the same names in the global namespace; not sure if we can containerize that
// For now, the types remain binary-compatible though
#ifdef DOUBLEPRECISION
    #define FLOAT double
    #define FLOAT4 double4
    #define FLOAT3 double3
    #define RSQRT rsqrt
    #define MIN fmin
    #define ItemSize 2
#else
    #define FLOAT float
    #define FLOAT4 float4
    #define FLOAT3 float3
    #define RSQRT rsqrtf
    #define MIN fminf
    #define ItemSize 1
#endif

#define posstruct FLOAT3
#ifdef COMPUTE_FOF_DENSITY
    #define accstruct FLOAT4
#else
    #define accstruct FLOAT3
#endif

#include "STimer.h"
#include "DeviceFunctions.h"
#include "IncludeGPUKernels.cuh"

// Provide a mechanism to call singlestep's STDLOG from this compilation unit
// TODO: consider writing to a new log file, as we do with the IO
void stdlog_hook(int verbosity, const char* str);
#define STDLOG(_verbosity,...) do { \
    char logstr[1024]; \
    sprintf(logstr, __VA_ARGS__);\
    stdlog_hook(_verbosity, logstr);\
} while(0)

#define assertf(_mytest,...) do { \
    if (!(_mytest)) { \
        STDLOG(0,"Failed Assertion: %s\n", #_mytest);\
        fprintf(stderr,"Failed Assertion: %s\n", #_mytest);\
        STDLOG(0,__VA_ARGS__);\
        assert(0==99); \
    }} while(0)


// Forward declarations that will get linked later
int set_core_affinity(int core_id);

#include "SetInteractionCollection.hh"

int NGPU = 0;                // Number of actual CUDA devices (not threads)
int NGPUQueues = 0;  // Number of actual work queues

// ===================  The Queue of Tasks =====================

/// We communicate to the queues with a simple FIFO queue. 
/// Tasks go in, and are read out by the QueueWatchers.
/// This class is the item that goes into that queue for
/// the inter-thread communication.
struct GPUQueueTask {
    void *task; ///< The pointer to the task data
    int type;   ///< == TASKTYPE_KILL signals to kill the Watcher
                ///< == TASKTYPE_SIC is for SetInteractionCollections
};

enum GPUQueueTaskType { TASKTYPE_KILL,
                        TASKTYPE_SIC, };

#include "tbb/concurrent_queue.h"
tbb::concurrent_bounded_queue<GPUQueueTask> *work_queues;



// =============== GPU Buffers =============================

/// The memory in each GPU is divided into a handful of equal sized
/// buffers, with matching pinned space on the host.  
/// Each buffer is paired with one thread.  Tasks will occupy one
/// buffer.  The memory allocations persist until the thread ends.

struct GPUBuffer {
    uint64 size;        ///< The size in bytes
    uint64 sizeWC;        ///< The size in bytes of the WC part
    uint64 sizeDef;        ///< The size in bytes of the non-WC part
    char * device;        ///< The device-side memory
    char * host;        ///< The host-side memory allocated as default
    char * hostWC;        ///< The host-side memory allocated as write combined
    volatile int ready;            ///< Have we finished initializing the CUDA stream for this buffer?
};

// GPUSetup() spins up one thread per GPU buffer
// Each will be passed this struct which contains information
// about which GPU to attach to, whether to use pinned memory, etc.
struct ThreadInfo {
    int thread_num;  // thread/buffer number
    int core;
    int queue;  // which queue are we watching?
    int UsePinnedGPUMemory;
    pthread_barrier_t *barrier; // this is a pointer because threads share barriers
    pthread_t thread;  // the thread itself
};

hipStream_t * DeviceStreams;        ///< The actual CUDA streams, one per thread
GPUBuffer * Buffers;                 ///< The pointers to the allocated memory
ThreadInfo *DeviceThreads;        ///< The threads!


// =============== Code to invoke execution on a SIC ============

// int CurrentGPU = 0;

void GPUPencilTask(void *, int);

/// This submits a single SetInteractionCollection to the GPU queue.
void SetInteractionCollection::GPUExecute(int blocking){
    // This method is invoked on a SIC in order to schedule it for execution.
    // At this point, we are directing work to individual GPUs.

    // Push to the NUMA-appropriate queue
    int q = ((j_low + j_high) / 2. / cpd) * NGPUQueues;
    if(j_low == j_high && j_high == cpd)
        q--;
    assert(q < NGPUQueues);

    Blocking = blocking;
    
    if(blocking){
        // We're running the task from the main thread;
        // make sure the GPU threads have finished the necessary CUDA setup
        while(!Buffers[0].ready){}
        
        // Send to buffer 0. No need to load balance when blocking!
        GPUPencilTask(this, 0);
    }
    else {
        GPUQueueTask t;
        t.type = TASKTYPE_SIC;
        t.task = (void *)this;
        work_queues[q].push(t);
    }
}


// ====================== GPU configuration =====================

/// Function to get the number of GPUs
extern "C"
// Use ./configure --with-max-gpus=1 to force 1 GPU
int GetNGPU(){
    int ngpu;
    checkCudaErrors(hipGetDeviceCount(&ngpu));
    if(ngpu > MAX_GPUS)
        ngpu = MAX_GPUS;
    return ngpu;
}

/// Function to get the amount of memory on the GPUs
extern "C" double GetDeviceMemory(){
    int ngpu;
    checkCudaErrors(hipGetDeviceCount(&ngpu));
    double mem = 1e99;
    for(int g = 0; g < ngpu; g++){
        hipDeviceProp_t p;
        checkCudaErrors(hipGetDeviceProperties(&p,g));
        double m = p.totalGlobalMem/1e9;
        mem = min(.8*m,mem);
    }
    return mem;
}


// =============== Setting up the GPUs =========================

// These are the sizes we allocate on the GPU/pinned memory
int MaxSinkBlocks, MaxSourceBlocks;
int MaxNSink, MaxNSource;
size_t MaxSinkSize, MaxSourceSize;

int init = 0;
int BPD;                 // Buffers per device

static volatile uint64 host_alloc_bytes;

void *QueueWatcher(void *);

// Here is the routine that is called to configure the GPU threads
// and then actually initiate the threads.

// We given the number of GPUs, buffers per GPU, and bytes per buffer.
// Also CPD and the number of particles.
// We use this to compute and return maximum task sizes.

/*
For the Pencil task, the average memory usage:

Memory per Block:
Sink pos and accel:  
        FLOAT3*NFBlockSize bytes of WC memory
        + accstruct*NFBlockSize bytes of Default memory
Source Pos: FLOAT3*NFBLockSize, but we expect 5x more SourceBlocks
SinkBlockParentPencil: An extra 4 bytes per block.

Sink Pencils: 4+(4+FLOAT)*WIDTH
Source Pencils: 4+4
    This is very small, typically <1%
    We can be conservative and assume this to be per block rather than per cell.
    This slightly neglects the fact that the sources need 4 boundary
    rows, but we just built in a little extra.

In summary:
Per SourceBlock: FLOAT3*NFBlockSize+8/WIDTH of WC memory
Per SinkBlock:   FLOAT3*NFBlockSize+4+FLOAT+4/WIDTH of WC memory
                 accstruct*NFBlockSize of default memory

Assuming that the number of SourceBlocks is WIDTHx SinkBlocks,
this sets the ratio of WC to default memory.  And we can then
return the maximum number of Source and Sink Blocks, so that
the upstream routine can size the SIC to match.

We ignore the fact that the Sink/Source Pencil count may be 
limited by the fraction of CPD**2

*/



/// This launches all of the GPU threads and associated control.
///
/// We will be running multiple GPU threads, usually a few per device.
/// Each thread can only do one task (e.g., a SIC) at a time, and 
/// it controls a single CUDA stream that guarantees sequential 
/// execution of the load, compute, and store cycle.
/// We have multiple threads per GPU so that load/store can be
/// overlapped with compute.
/// 
/// This routine is supplied with the size (in bytes) of each
/// GPU buffer, and it returns estimates on that basis of how
/// big the tasks can be.  This is used in the planning of tasks.
///
/// Note that we size this to NFRADIUS, which is allowed to be bigger
/// than P.NearFieldRadius.

extern "C" void GPUSetup(int cpd, uint64 MaxBufferSize, 
    int numberGPUs, int bufferperdevice, 
    int *ThreadCoreStart, int NThreadCores,
    int *GPUQueueAssignments,
    int *maxsinkblocks, int *maxsourceblocks,
    int UsePinnedGPUMemory) {

    if (init != 0) return;
    BPD = bufferperdevice;
    NGPU = numberGPUs;
    int NBuf = BPD*NGPU;

    // Determine the number of queues by the max assigned queue
    for (int i = 0; i < NGPU; i++){
        NGPUQueues = max(NGPUQueues, GPUQueueAssignments[i]+1);
    }
    STDLOG(1, "Using %d GPU work queues\n", NGPUQueues);

    // The following estimates are documented above, and assume sizeof(int)=4
    float BytesPerSourceBlockWC = sizeof(FLOAT)*3*NFBlockSize+8.0;
    float BytesPerSinkBlockWC = sizeof(FLOAT)*3*NFBlockSize
                    +(4.0+sizeof(FLOAT))*WIDTH+4.0;
    float BytesPerSinkBlockDef = sizeof(accstruct)*NFBlockSize;

    // Set the levels assuming WIDTH Sources per Sink
    float TotalBytesPerSinkBlock = BytesPerSinkBlockDef+
            BytesPerSinkBlockWC+WIDTH*BytesPerSourceBlockWC;
    STDLOG(2, "Bytes per Block = %5.1f+%5.1f+%d*%5.1f = %5.1f\n",
            BytesPerSinkBlockDef, BytesPerSinkBlockWC, WIDTH, BytesPerSourceBlockWC,
            TotalBytesPerSinkBlock);

    // We're splitting the Buffer into WC and Def memory
    float RatioDeftoAll = BytesPerSinkBlockDef/TotalBytesPerSinkBlock;
    assert(RatioDeftoAll<=1.0);  // Guard against screwup

    // This is how many blocks we'll allocate
    MaxSinkBlocks = (MaxBufferSize-1e5)/TotalBytesPerSinkBlock;
            // Remove 100KB for alignment factors
    MaxSourceBlocks = WIDTH*MaxSinkBlocks;
    *maxsinkblocks = MaxSinkBlocks;
    *maxsourceblocks = MaxSourceBlocks;
    // The number of particles corresponding to these
    MaxSinkSize     = NFBlockSize * MaxSinkBlocks;
    MaxSourceSize   = NFBlockSize * MaxSourceBlocks;

    STDLOG(2, "Planning for %d sink and %d source blocks, each %d particles\n",
            MaxSinkBlocks, MaxSourceBlocks, NFBlockSize);

    // And then we have storage for the Pencils.
    // Here we pessimistically assume one Pencil per SinkBlock.
    MaxNSink = MaxSinkBlocks;
    MaxNSource = MaxNSink;
    assertf(MaxNSink>2*cpd, "MaxNSink = %d is too small\n", MaxNSink);
    assertf(MaxNSource>(2+WIDTH)*cpd, "MaxNSource = %d is too small\n", MaxNSource);
    // Formally, this one could be slightly larger because of the boundary
    // rows.  However, we neglected this in the Memory estimate, so we'll
    // overflow our buffer if we put it in here.  Fortunately, 1 Pencil
    // per SinkBlock assumption is really very conservative.  Only likely
    // counter-case is very low PPC sims, where one would typically have
    // a small problem that fits very easily on the GPU (i.e., MaxBufferSize
    // will have led to a MaxNSink that is > CPD**2.


    DeviceStreams = new hipStream_t[NBuf];
    Buffers = new GPUBuffer[NBuf];
    DeviceThreads = new ThreadInfo[NBuf];
    work_queues = new tbb::concurrent_bounded_queue<GPUQueueTask>[NGPUQueues];
    
    // Start one thread per buffer
    
    // Assign threads to cores
    // TODO: make this automatic using libnuma
    //int n_socket = numa_available() == -1 ? 1 : (numa_max_node() + 1);
    //STDLOG(1, "Detected %d sockets/NUMA nodes\n", n_socket);
    //assertf(n_socket > 0, "n_socket %d less than 1\n", n_socket);

    if(UsePinnedGPUMemory < 0)
        UsePinnedGPUMemory = MaxSinkBlocks >= 10000;  // Pinning is slow, so for very small problems it's faster to use unpinned memory

    if(UsePinnedGPUMemory)
        STDLOG(1, "Allocating pinned memory\n");
    else
        STDLOG(1, "Allocating host-side memory, but not pinning because this is a small problem\n");

    pthread_barrier_t *thread_startup_barriers[NGPU];
    for(int g = 0; g < NBuf; g++){
        Buffers[g].size = MaxBufferSize;
        Buffers[g].sizeWC = Buffers[g].size*(1.0-RatioDeftoAll);
        Buffers[g].sizeDef = Buffers[g].size*RatioDeftoAll;
        Buffers[g].ready = 0;

        int core_start = ThreadCoreStart[g % NGPU];
        int core = -1;
        // If either the core start or the core count are invalid, do not bind this thread to a core
        if(core_start >= 0 && NThreadCores > 0)
            // cycle through cores, but keep each GPU within P.NGPUThreadCores of its starting core
            core = core_start + ((g/NGPU) % NThreadCores);

        if(g < NGPU){
            thread_startup_barriers[g] = new pthread_barrier_t;
            int p_ret = pthread_barrier_init(thread_startup_barriers[g], NULL, BPD);
            assertf(p_ret == 0, "pthread_barrier_init failed with value %d\n", p_ret);
        }
        
        ThreadInfo *info = DeviceThreads + g;
        info->thread_num = g;
        info->core = core;
        info->UsePinnedGPUMemory = UsePinnedGPUMemory;
        info->barrier = thread_startup_barriers[g%NGPU];
        info->queue = GPUQueueAssignments[g%NGPU];

        if(core >= 0)
            STDLOG(0, "GPU buffer thread %d (GPU %d) assigned to core %d, watching queue %d\n", g, g % NGPU, core, info->queue);
        else
            STDLOG(0, "GPU buffer thread %d (GPU %d) not bound to a core, watching queue %d\n", g, g % NGPU, info->queue);
        
        // Start one thread per buffer
        int p_retval = pthread_create(&(DeviceThreads[g].thread), NULL, QueueWatcher, info);
        assertf(p_retval == 0, "pthread_create failed with value %d\n", p_retval);

        host_alloc_bytes += Buffers[g].sizeWC + Buffers[g].sizeDef;
    }

    STDLOG(1, "Allocated %f MB host-side memory\n", host_alloc_bytes/1024./1024);

    init = 1;
}


/// This is the routine to turn off the GPU threads,
/// reset the device, and delete the host-side memory.

void GPUReset(){
    GPUQueueTask t;
    t.type = TASKTYPE_KILL;
    t.task = NULL;
    for(int i = 0; i < BPD*NGPU; i++)
        work_queues[DeviceThreads[i].queue].push(t);
    for(int i = 0; i < BPD*NGPU; i++)
        assert(pthread_join(DeviceThreads[i].thread, NULL) == 0);
    for(int i = 0; i < NGPUQueues; i++){
        assert(work_queues[i].empty());
    }

    hipDeviceReset();

    delete[] Buffers;
    delete[] DeviceStreams;
    delete[] DeviceThreads;
    delete[] work_queues;
}


// =============== Main code for the GPU thread =================

#define CudaAllocate(ptr,size) checkCudaErrors(hipMalloc((void **)&(ptr), size))

#define PinnedAllocate(ptr,size) if(UsePinnedGPUMemory) checkCudaErrors(hipHostAlloc((void **)&(ptr), size, hipHostMallocDefault)); \
                                    else assert(posix_memalign((void **)&(ptr), CACHE_LINE_SIZE, size) == 0);

#define WCAllocate(ptr,size) if(UsePinnedGPUMemory) checkCudaErrors(hipHostAlloc((void **)&(ptr), size, hipHostMallocWriteCombined)); \
                                    else assert(posix_memalign((void **)&(ptr), CACHE_LINE_SIZE, size) == 0);

#include "hip/hip_runtime_api.h"

/// Here is the top-level function for each GPU thread.
/// It allocates the space, then monitors a queue to get its work tasks
/// and then invoke the Task function.  When it receives the queue
/// signal to quit, it deletes its buffer memory.
///
/// A wrapper to GPUPencilTask that executes work units from the queue
/// Intended to be executed by several threads/buffers in parallel
/// Our original intention was to let the CUDA stream be our "queue"
/// but the copyback operation is blocking unless we pin memory
/// So this serves as a non-blocking queue.
///
/// By handling buffer memory here, we get a favorable NUMA situation.
/// However, it is also useful to have the thread running on the 
/// correct socket for its PCI slot, so that is passed in as well
/// to run set_core_affinity().

void *QueueWatcher(void *arg){
    ThreadInfo* info = (ThreadInfo *) arg;
    int assigned_device = info->thread_num;
    int n = assigned_device;                 // The buffer number
    int gpu = assigned_device % NGPU;        // The GPU device number
    int queue = info->queue;
    int assigned_core = info->core;
    if (assigned_core >= 0)
        set_core_affinity(assigned_core);
    int UsePinnedGPUMemory = info->UsePinnedGPUMemory;

    STDLOG(1,"Running GPU thread %d, core %d\n", n, assigned_core);

    checkCudaErrors(hipSetDevice(gpu));
    if (assigned_device < NGPU) {
        // Only run these the first time a GPU is seen
        checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
        checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
    }

    // Wait on the barrier: the "head" thread for each GPU must complete setting device flags before any CUDA operations can occur
    pthread_barrier_wait(info->barrier);
    STDLOG(1,"Barrier passed on stream %d\n", n);

    // Initiate the stream
    checkCudaErrors(hipStreamCreateWithFlags(&DeviceStreams[n], hipStreamNonBlocking));
    STDLOG(1,"GPU stream %d initiated\n", n);

    // Allocate CUDA memory
    CudaAllocate(Buffers[n].device,     Buffers[n].size);
    WCAllocate(Buffers[n].hostWC,       Buffers[n].sizeWC);
    PinnedAllocate(Buffers[n].host,     Buffers[n].sizeDef);
    STDLOG(1,"GPU thread %d memory allocated\n", n);
    // We make 2/3 of the host pinned memory WriteCombined, which is
    // good for sending data to the GPU.  The other 1/3 is normal, 
    // better for returning the data from the GPU.
    
#ifdef HAVE_LIBNUMA
    // Query the current NUMA node of the allocated buffers
    // We are using the move_pages function purely to query NUMA state, not move anything
    int page = -1, ret = 0;
    ret = move_pages(0, 1, (void **) &(Buffers[n].host), NULL, &page, 0);
    if(ret == 0)
        STDLOG(1, "Host buffer for GPU %d allocated on NUMA node %d on core %d\n", gpu, page, assigned_core);
    else
        STDLOG(1, "NUMA page query failed for GPU %d on core %d\n", gpu, assigned_core);
    ret = move_pages(0, 1, (void **) &(Buffers[n].hostWC), NULL, &page, 0);
    if(ret == 0)
        STDLOG(1, "Host write-combined buffer for GPU %d allocated on NUMA node %d on core %d\n", gpu, page, assigned_core);
    else
        STDLOG(1, "NUMA page query failed on host write-combined buffer for GPU %d on core %d\n", gpu, assigned_core);
#endif

    Buffers[n].ready = 1;

    // Main work loop: watch the queue
    while(true){
        GPUQueueTask item;
        work_queues[queue].pop(item);
        if (item.type == TASKTYPE_KILL)
            break;
        // Each thread (not work unit) is bound to a device

        if (item.type == TASKTYPE_SIC)
            GPUPencilTask(item.task, assigned_device);
    }

    STDLOG(1, "Received item signaling termination in GPU thread %d\n", n);
    
    // All done; make sure profiling info is sync'd
    checkCudaErrors(hipStreamSynchronize(DeviceStreams[assigned_device]));
    checkCudaErrors(hipProfilerStop());

    // Free our memory
    /* Leaking these saves some time, maybe a few seconds
    checkCudaErrors(hipFree(Buffers[n].device));
    if (UsePinnedGPUMemory) {
        // The following hipHostFree lines cause this error on summitdev
        // unless jsrun is used with "--smpiargs off"
        // "CUDA Hook Library: Failed to find symbol mem_find_dreg_entries, /dev/shm/lgarrison/abacus/singlestep/singlestep: undefined symbol: __PAMI_Invalidate_region"
        // Worst case, it's probably safe to comment out these lines and leak these, since we're about to exit anyway
        checkCudaErrors(hipHostFree(Buffers[n].hostWC));
        checkCudaErrors(hipHostFree(Buffers[n].host));
    } else {
        free(Buffers[n].hostWC);
        free(Buffers[n].host);
    }*/
    
    STDLOG(1, "Terminated GPU thread %d\n", n);

    if(assigned_device < NGPU)
        delete info->barrier;  // this was allocated in GPUSetup

    return NULL;
}

#undef CudaAllocate
#undef PinnedAllocate
#undef WCAllocate


// ================= Timing the GPUs ==================

// TODO: This may need some adjustment when there's more than one task type.
// Or maybe not?  


/// Timing the GPU is tricky because there are multiple threads 
/// issuing work to it.  So we keep a global counter of all of 
/// active threads, and run a timer when that count is non-zero.
/// That gives a global throughput number.

void CUDART_CB StartThroughputTimer(hipStream_t stream, hipError_t status, void *data){
    assert(pthread_mutex_lock(&SetInteractionCollection::GPUTimerMutex) == 0);
    SetInteractionCollection::ActiveThreads++;
    if (SetInteractionCollection::ActiveThreads == 1)
        SetInteractionCollection::GPUThroughputTimer.Start();
    assert(pthread_mutex_unlock(&SetInteractionCollection::GPUTimerMutex) == 0);
}

void CUDART_CB MarkCompleted( hipStream_t stream, hipError_t status, void *data){
#ifdef CUDADIRECT
    
    assert(pthread_mutex_lock(&SetInteractionCollection::GPUTimerMutex) == 0);
    SetInteractionCollection::ActiveThreads--;
    if (SetInteractionCollection::ActiveThreads == 0)
        SetInteractionCollection::GPUThroughputTimer.Stop();
    assert(pthread_mutex_unlock(&SetInteractionCollection::GPUTimerMutex) == 0);
#endif
}


// ==================  The specific tasks =======================

/// Counting the number of direct interactions performed
__device__ unsigned long long int DI;

__global__ void getDI(unsigned long long int * h_di){
    *h_di = DI;
}

// Here is the code that knows how to execute a single Pencil task.
#include "PencilTask.cu"
