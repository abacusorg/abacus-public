/// This is code for the Global Pencil-on-Pencil work.
///
/// In particular, the function GPUPencilTask receives a pointer
/// to a SetInteractionCollection and an assignment of a GPUBuffer
/// and Stream.
///
/// It must load the host-side data into Pinned Memory and then the GPU,
/// then execute the direct kernel, then copy the data back into the 
/// SIC-supplied memory.

// ======================== DeviceData =====================

/// This is the structure that is actually passed to the GPU,
/// containing all of the information from a SIC instance.

struct DeviceData{
    List3<FLOAT>    SinkSetPositions;
    List3<FLOAT>    SourceSetPositions;
    
    int *           SinkSetIdMax;
    accstruct *        SinkSetAccelerations;
    FLOAT	    b2;
    int             NSinkBlocks;
    int *           SinkBlockParentPencil;

    int *           SourceSetStart;
    int *           SourceSetCount;
    int             NSourceBlocks;

    int             NSinkSets;
    int             NSourceSets;
    int             InteractionCount;

    int *           SinkSourceInteractionList;
    FLOAT *         SinkSourceYOffset;
};

// Here's the GPU code that knows what to do with a DeviceData structure
#include "PencilKernel.cu"

// ================ Configure Buffer as DeviceData ==============

// Just a little thing to avoid compiler warnings
void SetPointer(char **p, char *val) { *p = val; }

// Some macros to help readability
#define   CudaConfig(ptr, size) SetPointer((char **)&ptr, buf.device+used_gpu); used_gpu+=(size/4096+1)*4096;

#define     WCConfig(ptr, size) SetPointer((char **)&ptr, buf.hostWC+used_hostWC); used_hostWC+=(size/4096+1)*4096;

#define PinnedConfig(ptr, size) SetPointer((char **)&ptr, buf.host+used_host); used_host+=(size/4096+1)*4096;


/// Given a Buffer, we want to set pointers in the DeviceData's that
/// divide it up to our needed vectors.
void ConfigureBufferAsDeviceData(GPUBuffer &buf, 
	DeviceData &gpu, DeviceData &pinned) {
    uint64 used_gpu = 0;
    uint64 used_host = 0;
    uint64 used_hostWC = 0;

    // Allocate GPU-side memory
    CudaConfig(gpu.SinkSetIdMax,              sizeof(int) * MaxNSink);
    CudaConfig(gpu.SourceSetStart,            sizeof(int) * MaxNSource);
    CudaConfig(gpu.SourceSetCount,            sizeof(int) * MaxNSource);
    CudaConfig(gpu.SinkSourceInteractionList, sizeof(int) * MaxNSink * WIDTH);
    CudaConfig(gpu.SinkSourceYOffset,         sizeof(FLOAT) * MaxNSink*WIDTH);
    CudaConfig(gpu.SinkBlockParentPencil,     sizeof(int) * MaxSinkBlocks);
    CudaConfig(gpu.SinkSetPositions.X,        sizeof(FLOAT) * MaxSinkSize);
    CudaConfig(gpu.SinkSetPositions.Y,        sizeof(FLOAT) * MaxSinkSize);
    CudaConfig(gpu.SinkSetPositions.Z,        sizeof(FLOAT) * MaxSinkSize);
    CudaConfig(gpu.SinkSetAccelerations,      sizeof(accstruct) * MaxSinkSize);
    CudaConfig(gpu.SourceSetPositions.X,      sizeof(FLOAT) * MaxSourceSize);
    CudaConfig(gpu.SourceSetPositions.Y,      sizeof(FLOAT) * MaxSourceSize);
    CudaConfig(gpu.SourceSetPositions.Z,      sizeof(FLOAT) * MaxSourceSize);
    assertf(used_gpu<buf.size, "Configuration of Buffer requesting %ld bytes on device, but only %ld available\n", used_gpu, buf.size);   // Check that we didn't overflow

    // Allocate host-side buffers
    WCConfig(pinned.SinkSetIdMax,              sizeof(int) * MaxNSink);
    WCConfig(pinned.SourceSetStart,            sizeof(int) * MaxNSource);
    WCConfig(pinned.SourceSetCount,            sizeof(int) * MaxNSource);
    WCConfig(pinned.SinkSourceInteractionList, sizeof(int) * MaxNSink * WIDTH);
    WCConfig(pinned.SinkSourceYOffset,         sizeof(FLOAT) * MaxNSink * WIDTH);
    WCConfig(pinned.SinkBlockParentPencil,     sizeof(int) * MaxSinkBlocks);
    WCConfig(pinned.SinkSetPositions.X,        sizeof(FLOAT) * MaxSinkSize);
    WCConfig(pinned.SinkSetPositions.Y,        sizeof(FLOAT) * MaxSinkSize);
    WCConfig(pinned.SinkSetPositions.Z,        sizeof(FLOAT) * MaxSinkSize);
    PinnedConfig(pinned.SinkSetAccelerations,  sizeof(accstruct) * MaxSinkSize);
    WCConfig(pinned.SourceSetPositions.X,      sizeof(FLOAT) * MaxSourceSize);
    WCConfig(pinned.SourceSetPositions.Y,      sizeof(FLOAT) * MaxSourceSize);
    WCConfig(pinned.SourceSetPositions.Z,      sizeof(FLOAT) * MaxSourceSize);
    assert(used_host<buf.sizeDef);
    assert(used_hostWC<buf.sizeWC);
    assertf(used_host<buf.sizeDef, "Configuration of Buffer requesting %ld bytes on host Def, but only %ld available\n", used_host, buf.sizeDef);   // Check that we didn't overflow
    assertf(used_hostWC<buf.sizeWC, "Configuration of Buffer requesting %ld bytes on host WC, but only %ld available\n", used_host, buf.sizeWC);   // Check that we didn't overflow
    return;
}

#undef CudaConfig
#undef WCConfig
#undef PinnedConfig

// ============= GPUPencilTask: executing one SetInteractionCollection =======

// Space-saving macros.  We rely on the idea that the name of a field
// is the same in the SetInteractionCollection and the DeviceData structures.

// This copies from the SIC task to the Pinned host-side DeviceData,
// and then to the GPU.
#define CopyToGPU(name,size)\
    thissize = size;\
    memcpy(PinnedBuffer.name, task->name, thissize);\
    checkCudaErrors(hipMemcpyAsync(StreamData.name, PinnedBuffer.name, thissize, hipMemcpyHostToDevice, DeviceStreams[g]));\
    size_to_gpu += thissize

// For the particle lists, we've loaded the pinned memory separately,
// so we only need to copy to the GPU
#define CopyListToGPU(name,d,size)\
    thissize = size;\
    checkCudaErrors(hipMemcpyAsync(StreamData.name.d, PinnedBuffer.name.d, thissize, hipMemcpyHostToDevice, DeviceStreams[g]));\
    size_to_gpu += thissize

// This copies back from the GPU to Pinned memory
#define CopyFromGPU(name,size)\
    thissize = size;\
    checkCudaErrors(hipMemcpyAsync(PinnedBuffer.name, StreamData.name, thissize, hipMemcpyDeviceToHost, DeviceStreams[g]));\
    size_from_gpu += thissize



/// This routine is invoked on a single SetInteractionCollection.
///
/// It must copy the relevant contents to a DeviceData instance in
/// the pinned memory, then copy that to the matching DeviceData 
/// instance on the GPU.  Then it invokes the GPU kernel, with
/// the GPU-side DeviceData as the argument.  Then it copies the
/// partial accelerations back from the GPU to pinned memory, and
/// then to space that was allocated in the SIC.  Then marks the SIC
/// as completed.
///
/// Importantly, this task has substantial host-side work too, because
/// the SIC does not contain the actual particle data, but only the 
/// instructions on where to find it.  So we must invoke the SinkPencilPlan's
/// and SourcePencilPlan's to actually copy the data from the slab locations
/// into the Pinned memory.

void GPUPencilTask(void *item, int g){
#ifdef CUDADIRECT
    // Given a buffer/stream number g, and a pointer to our task, do it.
    SetInteractionCollection *task = (SetInteractionCollection *)item;
    task->DeviceThreadTimer.Start();
    task->LaunchDeviceKernels.Start();
    task->AssignedDevice = g;
    checkCudaErrors(hipSetDevice(g%NGPU));  // only needed for blocking directs

    DeviceData StreamData, PinnedBuffer;

    ConfigureBufferAsDeviceData(Buffers[g], StreamData, PinnedBuffer);

    //Schedule data copy to GPU
    size_t thissize = 0;
    uint64 size_to_gpu = 0, size_from_gpu = 0;
    StartThroughputTimer(DeviceStreams[g], hipSuccess, (void *) task);
    // Zero the device-side accelerations
    checkCudaErrors(hipMemsetAsync(StreamData.SinkSetAccelerations,
                0 , sizeof(accstruct) * NFBlockSize * task->NSinkBlocks, DeviceStreams[g]));
    StreamData.InteractionCount = task->InteractionCount;
    StreamData.b2 = task->b2;

    // Need to load the particles to the PinnedBuffer.
    // Copy the sinks into position
    task->LaunchDeviceKernels.Stop();
    task->FillSinks.Start();
    for (int j=0; j<task->NSinkSets; j++) {
        task->SinkPlan[j].copy_into_pinned_memory(PinnedBuffer.SinkSetPositions, task->SinkSetStart[j], task->SinkSetCount[j]);
    }
    task->FillSinks.Stop();
    task->LaunchDeviceKernels.Start();
    
    // Now copy these to the GPU
    CopyListToGPU(SinkSetPositions, X, sizeof(FLOAT) * task->NSinkBlocks * NFBlockSize);
    CopyListToGPU(SinkSetPositions, Y, sizeof(FLOAT) * task->NSinkBlocks * NFBlockSize);
    CopyListToGPU(SinkSetPositions, Z, sizeof(FLOAT) * task->NSinkBlocks * NFBlockSize);

    task->LaunchDeviceKernels.Stop();

    // Repeat this with the sources
    task->FillSources.Start();
    for (int j=0; j<task->NSourceSets; j++) {
        task->SourcePlan[j].copy_into_pinned_memory(PinnedBuffer.SourceSetPositions, task->SourceSetStart[j], task->SourceSetCount[j]);
    }
    task->FillSources.Stop();
    task->LaunchDeviceKernels.Start();
    
    CopyListToGPU(SourceSetPositions, X, sizeof(FLOAT) * task->NSourceBlocks * NFBlockSize);
    CopyListToGPU(SourceSetPositions, Y, sizeof(FLOAT) * task->NSourceBlocks * NFBlockSize);
    CopyListToGPU(SourceSetPositions, Z, sizeof(FLOAT) * task->NSourceBlocks * NFBlockSize);

    // Now copy other information from the SIC to the GPU
    CopyToGPU(SinkSetIdMax,                 sizeof(int)*task->NSinkSets);
    CopyToGPU(SinkBlockParentPencil,        sizeof(int)*task->NSinkBlocks);
    CopyToGPU(SourceSetStart,               sizeof(int)*task->NSourceSets);
    CopyToGPU(SourceSetCount,               sizeof(int)*task->NSourceSets);
    CopyToGPU(SinkSourceInteractionList,    sizeof(int)*task->InteractionCount);
    CopyToGPU(SinkSourceYOffset,            sizeof(FLOAT)*task->InteractionCount);

    // Run the GPU kernel for this Interaction collection!
    dim3 dimGrid(task->NSinkBlocks);
    dim3 dimBlock(NFBlockSize);
    ComputeDirects<<<dimGrid,dimBlock,0,DeviceStreams[g]>>>(StreamData,task->eps);
    // Control won't return until it's done
    
    // Copy back results from GPU
    // If the memory is unpinned, this is blocking
    CopyFromGPU(SinkSetAccelerations, sizeof(accstruct) * NFBlockSize * task->NSinkBlocks);
    
    task->bytes_to_device = size_to_gpu;
    task->bytes_from_device = size_from_gpu;
    task->LaunchDeviceKernels.Stop();

    // It's safe for the next work unit to wipe the host PinnedBuffer once everything's been transferred
    // We could allow sink/source copying while waiting for accels, but we'd need a signalling mechanism from the callback
    task->WaitForResult.Start();
    checkCudaErrors(hipStreamSynchronize(DeviceStreams[g]));
    task->WaitForResult.Stop();

    // Now copy the data from Pinned back to the SIC buffer
    task->CopyAccelFromPinned.Start();
    // SetInteractionCollection * task = (SetInteractionCollection *) data;
    memcpy(task->SinkSetAccelerations, PinnedBuffer.SinkSetAccelerations, sizeof(accstruct) * NFBlockSize * task->NSinkBlocks);

    // Declare victory!
    task->SetCompleted();
    MarkCompleted(DeviceStreams[g], hipSuccess, (void *) task);
    task->CopyAccelFromPinned.Stop();
    task->DeviceThreadTimer.Stop();
#endif
}

#undef CopyToGPU
#undef CopyListToGPU
#undef CopyFromGPU
