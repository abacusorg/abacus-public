#include "hip/hip_runtime.h"
__global__ void ComputeDirects(DeviceData d, FLOAT eps2){

    __shared__ FLOAT SourceCacheX[NFBlockSize];
    __shared__ FLOAT SourceCacheY[NFBlockSize];
    __shared__ FLOAT SourceCacheZ[NFBlockSize];
    

    int id = blockDim.x*blockIdx.x + threadIdx.x;
    int myDI = 0;

    FLOAT sinkX, sinkY, sinkZ;
    int sinkIdx = d.SinkBlockParentPencil[blockIdx.x];
    if(id < d.SinkSetStart[sinkIdx] + d.SinkSetCount[sinkIdx]){
        sinkX = d.SinkSetPositions.X[id];
        sinkY = d.SinkSetPositions.Y[id];
        sinkZ = d.SinkSetPositions.Z[id];
    }else{
         sinkX =0;
         sinkY =0;
         sinkZ =0;
    }


    FLOAT3 a = {(FLOAT) 0.0,(FLOAT) 0.0,(FLOAT) 0.0};

    #ifdef DIRECTSPLINE
    eps2 = RSQRT(eps2);  // Direct spline uses 1/eps instead of eps^2
    #elif defined DIRECTCUBIC
    eps2 = eps2*eps2*RSQRT(eps2); // Direct cubic uses eps^3 instead of eps^2
    #endif
    
    int InteractionStart = sinkIdx * WIDTH;
    int InteractionMax =  InteractionStart + WIDTH;

    #pragma unroll
    for(int c = InteractionStart; c < InteractionMax; c++){
        int sourceIdx = d.SinkSourceInteractionList[c];
        int sourceStart = d.SourceSetStart[sourceIdx];
        int sourceCount = d.SourceSetCount[sourceIdx];
        int nB = sourceCount/NFBlockSize;

        for(int b = 0; b < nB; b+=1){
            int idx = sourceStart + b*NFBlockSize + threadIdx.x;
            SourceCacheX[threadIdx.x] = d.SourceSetPositions.X[idx];
            SourceCacheY[threadIdx.x] = d.SourceSetPositions.Y[idx];
            SourceCacheZ[threadIdx.x] = d.SourceSetPositions.Z[idx];
            __syncthreads();
            
            myDI += NFBlockSize;
            FullDirectTile( SourceCacheX, SourceCacheY, SourceCacheZ,
                    &sinkX, &sinkY, &sinkZ,
                    &(a.x),&(a.y),&(a.z),
                    &eps2);
            __syncthreads();

        }

        int remaining = sourceCount%NFBlockSize;

        if(threadIdx.x < remaining){
            int idx = sourceStart + nB*NFBlockSize + threadIdx.x;
            SourceCacheX[threadIdx.x] = d.SourceSetPositions.X[idx];
            SourceCacheY[threadIdx.x] = d.SourceSetPositions.Y[idx];
            SourceCacheZ[threadIdx.x] = d.SourceSetPositions.Z[idx];
        }
        __syncthreads();
        
        myDI += remaining;
        PartialDirectTile(SourceCacheX, SourceCacheY, SourceCacheZ,
                &sinkX, &sinkY, &sinkZ,
                &(a.x),&(a.y),&(a.z),
                &eps2, remaining);
        __syncthreads();
    }

    if(id < d.SinkSetStart[sinkIdx] + d.SinkSetCount[sinkIdx]){
        assert(isfinite(a.x));
        assert(isfinite(a.y));
        assert(isfinite(a.z));
        //atomicAdd(&(d.SinkSetAccelerations[id].x),a.x);
        //atomicAdd(&(d.SinkSetAccelerations[id].y),a.y);
        //atomicAdd(&(d.SinkSetAccelerations[id].z),a.z);
        d.SinkSetAccelerations[id] = a;
        atomicAdd(&DI, myDI);
    }

}
